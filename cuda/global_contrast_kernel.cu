#include "hip/hip_runtime.h"
// #include <torch/extension.h>
#include <torch/types.h>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>


#include <torch/types.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// #include "TimingGPU.cuh"

// hyper parameter 
const long BLOCK_SIZE = 32;

namespace global_contrast_kernel{

template <typename scalar_t> 
__global__ void forward(
    const torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> feature,
    torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> output
) {
    long col = threadIdx.x + blockIdx.x * blockDim.x;
    long row = threadIdx.y + blockIdx.y * blockDim.y;
    const long B = feature.size(0);
    const long C = feature.size(1);
    const long W = feature.size(2);
    const long H = feature.size(3);
 
    if(col >= W || row >= H)
        return;
    
    scalar_t dis = 0.0f;
    for (auto i=0 ; i<B ; i++){
        for (auto j=0 ; j<C ; j++){
            for (auto _w=0 ; _w<W ; _w++){
                for (auto _h=0 ; _h<H ; _h++){
                    scalar_t diff = feature[i][j][col][row] - feature[i][j][_w][_h];
                    dis += diff * diff;
                }
            }
        }
        output[i][0][col][row] = dis;
        dis = 0.0f;
    }

    __syncthreads();
}

template <typename scalar_t> 
__global__ void forward_split(
    const torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> feature,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> sum,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> square,
    torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> output
) {
    long col = threadIdx.x + blockIdx.x * blockDim.x;
    long row = threadIdx.y + blockIdx.y * blockDim.y;
    const long B = feature.size(0);
    const long C = feature.size(1);
    const long W = feature.size(2);
    const long H = feature.size(3);

    if(col >= W || row >= H)
        return;

    scalar_t dis = 0.0f, ft = 0.0f;
    for (auto i=0 ; i<B ; i++){
        for (auto j=0 ; j<C ; j++){
            ft = feature[i][j][col][row];
            dis += ft * ft + (square[i][j] - ft * sum[i][j] * 2.0f) / W / H; 
        }
        output[i][0][col][row] = dis / C;
        dis = 0.0f;
    }

    __syncthreads();
}

template <typename scalar_t>
__global__ void _calcSum_forward(
    const torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> feature,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> sum
) {

    long bs = threadIdx.x + blockIdx.x * blockDim.x;
    long ch = threadIdx.y + blockIdx.y * blockDim.y;
    const long B = feature.size(0);
    const long C = feature.size(1);
    const long W = feature.size(2);
    const long H = feature.size(3);

    if(bs >= B || ch >= C)
        return;

    scalar_t sm = 0.0f;
    for(auto i=0 ; i<W ; i++){
        for(auto j=0 ; j<H ; j++){
            sm += feature[bs][ch][i][j];
        }
    }
    sum[bs][ch] = sm;
    
    __syncthreads();

}

template <typename scalar_t>
__global__ void _calcSquare_forward(
    const torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> feature,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> square
) {

    long bs = threadIdx.x + blockIdx.x * blockDim.x;
    long ch = threadIdx.y + blockIdx.y * blockDim.y;
    const long B = feature.size(0);
    const long C = feature.size(1);
    const long W = feature.size(2);
    const long H = feature.size(3);

    if(bs >= B || ch >= C)
        return;

    scalar_t sq = 0.0f, temp = 0.0f;
    for(auto i=0 ; i<W ; i++){
        for(auto j=0 ; j<H ; j++){
            temp = feature[bs][ch][i][j];
            sq += temp * temp;
        }
    }
    square[bs][ch] = sq;

    __syncthreads();

}

template <typename scalar_t>
__global__ void _calcSum_backward(
    const torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> feature,
    torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> d_feature
) {

    long bs = threadIdx.x + blockIdx.x * blockDim.x;
    long ch = threadIdx.y + blockIdx.y * blockDim.y;
    const long B = feature.size(0);
    const long C = feature.size(1);
    const long W = feature.size(2);
    const long H = feature.size(3);

    if(bs >= B || ch >= C)
        return;

    scalar_t tmp = 0.0f;
    for(auto i=0 ; i<W ; i++){
        for(auto j=0 ; j<H ; j++){
            tmp += feature[bs][ch][i][j];
        }
    }
    tmp = (-1) * tmp / W / H;
    for(auto i=0 ; i<W ; i++){
        for(auto j=0 ; j<H ; j++){
            d_feature[bs][ch][i][j] = tmp;
        }
    }
    __syncthreads();

}

template <typename scalar_t>
__global__ void _calcGrad(
    const torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> grad,
    const torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> feature,
    torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> d_feature
) {

    long col = threadIdx.x + blockIdx.x * blockDim.x;
    long row = threadIdx.y + blockIdx.y * blockDim.y;
    const long B = feature.size(0);
    const long C = feature.size(1);
    const long W = feature.size(2);
    const long H = feature.size(3);

    if(col >= W || row >= H)
        return;

    for (auto i=0 ; i<B ; i++){
        for (auto j=0 ; j<C ; j++){
            d_feature[i][j][col][row] = (feature[i][j][col][row] + d_feature[i][j][col][row]) * 4.0f * grad[i][0][col][row];
        }
    }
    __syncthreads();

}

template <typename scalar_t> 
__global__ void backward(
    const torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> grad,
    const torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> feature,
    torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> d_feature
) {

    long col = threadIdx.x + blockIdx.x * blockDim.x;
    long row = threadIdx.y + blockIdx.y * blockDim.y;
    const long B = feature.size(0);
    const long C = feature.size(1);
    const long W = feature.size(2);
    const long H = feature.size(3);

    if(col >= W || row >= H)
        return;

    scalar_t tmp = 0.0f;
    for (auto i=0 ; i<B ; i++){
        for (auto j=0 ; j<C ; j++){
            for (auto _w=0 ; _w<W ; _w++){
                for (auto _h=0 ; _h<H ; _h++){
                    tmp += feature[i][j][_w][_h];
                }
            }
            d_feature[i][j][col][row] = (feature[i][j][col][row] - tmp / W / H ) * 4.0f * grad[i][0][col][row];
            tmp = 0.0f;
        }
    }
    
    __syncthreads();
}

}

torch::Tensor global_contrast_cuda_forward(
    const torch::Tensor& feature
) {

    hipSetDevice(feature.get_device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    const long B = feature.size(0);
    const long C = feature.size(1);
    const long W = feature.size(2);
    const long H = feature.size(3);

    // allocate output tensor
    auto output = torch::zeros({B, 1, W, H}, feature.options());

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((W + blockSize.x - 1) / blockSize.x, 
        (H + blockSize.y - 1) / blockSize.y);

    AT_DISPATCH_FLOATING_TYPES(feature.type(), "global_contrast_cuda_forward", ([&]{
        global_contrast_kernel::forward <scalar_t><<< gridSize, blockSize, 0, stream>>>(
            feature.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>(),
            output.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>()
        );
    }));
    hipDeviceSynchronize();

    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        std::cout << err <<std::endl;
        
    return output;
}

torch::Tensor global_contrast_cuda_forward_split(
    const torch::Tensor& feature
) {

    hipSetDevice(feature.get_device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    const long B = feature.size(0);
    const long C = feature.size(1);
    const long W = feature.size(2);
    const long H = feature.size(3);

    // allocate output tensor
    auto output = torch::zeros({B, 1, W, H}, feature.options());
    auto sum = torch::zeros({B, C}, feature.options());
    auto square = torch::zeros({B, C}, feature.options());

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((W + blockSize.x - 1) / blockSize.x, 
        (H + blockSize.y - 1) / blockSize.y);

    AT_DISPATCH_FLOATING_TYPES(feature.type(), "_calcSum_forward", ([&]{
        global_contrast_kernel::_calcSum_forward <scalar_t><<< gridSize, blockSize, 0, stream>>>(
            feature.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>(),
            sum.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>()
        );
    }));

    hipDeviceSynchronize();

    AT_DISPATCH_FLOATING_TYPES(feature.type(), "_calcSquare_forward", ([&]{
        global_contrast_kernel::_calcSquare_forward <scalar_t><<< gridSize, blockSize, 0, stream>>>(
            feature.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>(),
            square.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>()
        );
    }));

    hipDeviceSynchronize();

    AT_DISPATCH_FLOATING_TYPES(feature.type(), "forward_split", ([&]{
        global_contrast_kernel::forward_split <scalar_t><<< gridSize, blockSize, 0, stream>>>(
            feature.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>(),
            sum.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            square.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            output.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>()
        );
    }));

    hipDeviceSynchronize();

    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        std::cout << err <<std::endl;
    
    return output;
}


torch::Tensor global_contrast_cuda_backward(
    const torch::Tensor& grad,
    const torch::Tensor& feature
) {

    hipSetDevice(feature.get_device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    const long B = feature.size(0);
    const long C = feature.size(1);
    const long W = feature.size(2);
    const long H = feature.size(3);

    // allocate output tensor
    auto d_feature = torch::zeros({B, C, W, H}, feature.options());

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((W + blockSize.x - 1) / blockSize.x, 
        (H + blockSize.y - 1) / blockSize.y);

    AT_DISPATCH_FLOATING_TYPES(feature.type(), "global_contrast_cuda_backward", ([&]{
        global_contrast_kernel::backward <scalar_t><<< gridSize, blockSize>>>(
            grad.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>(),
            feature.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>(),
            d_feature.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>()
        );
    }));
    
    return d_feature;
}


torch::Tensor global_contrast_cuda_backward_split(
    const torch::Tensor& grad,
    const torch::Tensor& feature
) {

    hipSetDevice(feature.get_device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    const long B = feature.size(0);
    const long C = feature.size(1);
    const long W = feature.size(2);
    const long H = feature.size(3);

    // allocate output tensor
    auto d_feature = torch::zeros({B, C, W, H}, feature.options());

    dim3 blockSize(4, 4);
    dim3 gridSize((B + blockSize.x - 1) / blockSize.x, 
        (C + blockSize.y - 1) / blockSize.y);

    AT_DISPATCH_FLOATING_TYPES(feature.type(), "global_contrast_cuda_backward", ([&]{
        global_contrast_kernel::_calcSum_backward <scalar_t> <<< gridSize, blockSize, 0, stream>>>(
            feature.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>(),
            d_feature.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>()
        );
    }));


    blockSize = dim3 (BLOCK_SIZE, BLOCK_SIZE);
    gridSize = dim3((W + blockSize.x - 1) / blockSize.x, 
        (H + blockSize.y - 1) / blockSize.y);

    AT_DISPATCH_FLOATING_TYPES(feature.type(), "global_contrast_cuda_backward", ([&]{
        global_contrast_kernel::_calcGrad<scalar_t> <<< gridSize, blockSize, 0, stream>>>(
            grad.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>(),
            feature.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>(),
            d_feature.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>()
        );
    }));

    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        std::cout << err <<std::endl;

    return d_feature;
}

/*
gettimeofday( &end, NULL );
long  timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
gettimeofday( &start, NULL );
printf("The first stage time is %ld us\n", timeuse);

...

gettimeofday( &end, NULL );
timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
gettimeofday( &start, NULL );
printf("The second stage time is %ld us\n", timeuse);
*/
